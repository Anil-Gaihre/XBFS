#include <iostream>
#include "graph.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>
#include <list>
#include <vector>
#include "wtime.h"
#include <assert.h>
#include <fstream>
#include <math.h>

#include "bfs_adaptiveFQ_wb_async.cuh"

using namespace std;
int max_level(long int* arr, int n){
    long int val = arr[0];
    int idx = 0;
    for(int i = 1; i < n; i++){
        if(arr[i]>val){
            val = arr[i];
            idx = i;
        }
    }
    return idx;
}
int main(int args, char **argv)
{
	//typedef char data_t;//typedef int data_t;
    typedef int param_t;
    typedef long int index_t;
	std::cout<<"Input: ./exe beg csr weight BlockDim GridDim gpu_id results alpha src workload padding\n";
	if(args!=12){std::cout<<"Wrong input\n"; return -1;}
	
	const char *beg_file=argv[1];
	const char *csr_file=argv[2];
	const char *weight_file=argv[3];
    	
	param_t BlockDim = atoi(argv[4]);//int BlockDim = atoi(argv[1]);
    param_t GridDim = atoi(argv[5]);//int GridDim = atoi(argv[2]);
    //index_t total = atoi(argv[6]);//int iterNum = atoi(argv[3]);
    param_t id = atoi(argv[6]);
    const char *filename = argv[7];
    float alpha = (float)atof(argv[8]);
    param_t src = atoi(argv[9]);
    string dataset = argv[10];
    param_t padding = atoi(argv[11]);
    std::cout<<"alpha = "<<alpha<<"\n";
	//template <file_vertex_t, file_index_t, file_weight_t
	//new_vertex_t, new_index_t, new_weight_t>
	graph<int, long, int, int, long, char>
	*ginst = new graph
	<int, long, int, int, long, char>
	(beg_file,csr_file,weight_file);
   
    ofstream file(filename, ios::app);
    ofstream file1("Hybrid_iteration.csv", ios::app);
    //ofstream file(filename);
/*    ofstream edge("edge-list");
    for(int i = 0; i < ginst->vert_count; i++){
        edge<<i<<"'s neighor list: ";
        for(long int j = ginst->beg_pos[i]; j < ginst->beg_pos[i+1]; j++)
            edge<<ginst->csr[j]<<" ";
        edge<<"\n";
    }
*/    
    //std::cout<<"64842's degree: "<<ginst->beg_pos[64843]-ginst->beg_pos[64842]<<"\n";
    //std::cout<<"64842's beg_pos: "<<ginst->beg_pos[64842]<<" end_pos: "<<ginst->beg_pos[64843]<<"\n";
    //for(int i = ginst->beg_pos[64842]; i < ginst->beg_pos[64842+1]; i++)
    //    if(ginst->csr[i]==957364)
    //        std::cout<<"find!!!!!\n";
    //    std::cout<<ginst->csr[i]<<" ";
    //You can implement your single threaded graph algorithm here.
    //like BFS, SSSP, PageRank and etc.
    int* degree = new int[3];
    memset(degree, 0, 3 * sizeof(int));
    std::cout<<"Vertex classification based on the degree: \n";
    for(int i = 0; i < ginst->vert_count; i++){
        if(ginst->beg_pos[i+1] - ginst->beg_pos[i]<=32)
            degree[0]++;
        else if((ginst->beg_pos[i+1] - ginst->beg_pos[i])>32 && (ginst->beg_pos[i+1] - ginst->beg_pos[i])<=256){
            degree[1]++;
        }
        else if(ginst->beg_pos[i+1] - ginst->beg_pos[i]>256){
            degree[2]++;
        }
    }
    std::cout<<"small degree counts: "<<degree[0]<<" "<<"medium degree counts: "<<degree[1]<<" "<<"large degree counts: "<<degree[2];
    param_t init = 0;
    //for(int i = 0; i < ginst->vert_count; i++)
    //{
    //    if(ginst->beg_pos[i]>0){
    //       init = i - 1;
    //       break;
    //    }
    //}
    init = src;
    //std::cout<<"first 10 vertices' degree: \n";
    //for(int i = 0; i < 10; i++){
    //    std::cout<<ginst->beg_pos[i+1] - ginst->beg_pos[i]<<" ";
    //}
    std::cout<<"\nStarting BFS graph traversal from node "<<init<<"......\n";    
    int k = 0;
    param_t* status = new param_t[ginst->vert_count];
    for(int i = 0; i < ginst->vert_count; i++){
        status[i] = -1;
    }
    std::cout<<"The status array: ";
    //for(int i = 0; i < ginst->vert_count; i++)
    //    std::cout<<status[i]<<" ";
    bool isdone = false;
    status[init] = 0;
   //BFS traverse from level k to level k+1
    for(k = 0; isdone != true; k++){
        isdone = true;
        for(int i = 0; i < ginst->vert_count; i++){
            if(status[i]==k){
               for(long int j = ginst->beg_pos[i]; j < ginst->beg_pos[i+1]; j++){
                   if(status[ginst->csr[j]]==-1){
                      isdone = false;
                      status[ginst->csr[j]] = k+1;
                   }
               }
            }
        }
    }  
    std::cout<<"\nk = "<<k<<"\n";
    int invalid = 0;
    for(int i = 0; i < ginst->vert_count; i++){
        if(status[i]==-1)
            invalid++;
    //    if(status[i]==k-1)
    //    std::cout<<status[i]<<" ";
    }
    std::cout<<"\n#of remaining -1: "<<invalid<<"\n";
    std::cout<<"\nstarting processing the status array on GPU....\n";
    hipSetDevice(id);
    double* iter = new double[k];
    memset(iter, 0, (k)*sizeof(double));
    float* ratio = new float[k];
    memset(ratio, 0, (k)*sizeof(float));
    //param_t* front = new param_t[k];
    //memset(front, 0, (k)*sizeof(param_t));
    //for(int i = 0; i < k; i++){
    //    for(int j = 0; j < ginst->vert_count; j++){
    //        if(status[j]==i){
    //            front[status[j]]++;
    //        }
    //    }
    //}
    //std::cout<<"number of frontiers in each level: ";
    //for(int i = 0; i < k; i++)
    //    std::cout<<front[i]<<" ";
    //index_t* edge = new index_t[k];
    //memset(edge, 0, (k)*sizeof(index_t));
    //for(int i = 0; i < ginst->vert_count; i++){
    //    for(int j = 0; j < k; j++){
    //        if(status[i]==j){
    //            edge[j] += ginst->beg_pos[i+1] - ginst->beg_pos[i];
    //        }
    //    }
    //}
    //std::cout<<"\n number of edges in each level: ";
    //for(int i = 0; i < k; i++)
    //    std::cout<<edge[i]<<" ";
    //int temp2 = max_level(edge, k) - 1;
    //int adaptive = temp2;
    //std::cout<<"\nadaptive = "<<adaptive<<"\n";
    //for(int i = 0; i < temp2; i++)
    //    std::cout<<"adaptive-1/="<<(float)edge[i+1]/(float)edge[i]<<"   ";
    int adaptive=0;
    std::cout<<"The average degree of the graph is: "<<ginst->beg_pos[ginst->vert_count]/ginst->vert_count;
//================================================= 1. Atomic-based version start... ===========================================
    index_t* beg_cu;
    H_ERR(hipMalloc((void **)&beg_cu, sizeof(index_t)*(ginst->vert_count+1)));
    H_ERR(hipMemcpy(beg_cu, ginst->beg_pos, sizeof(index_t)*(ginst->vert_count+1), hipMemcpyHostToDevice));
    param_t* csr_cu;
    H_ERR(hipMalloc((void **)&csr_cu, sizeof(param_t)*(ginst->beg_pos[ginst->vert_count])));
    H_ERR(hipMemcpy(csr_cu, ginst->csr, sizeof(param_t)*(ginst->beg_pos[ginst->vert_count]), hipMemcpyHostToDevice));
    
    param_t* status_cpp;
    H_ERR(hipMalloc((void **)&status_cpp, sizeof(param_t)*(ginst->vert_count)));
    H_ERR(hipMemcpy(status_cpp, status, sizeof(param_t)*(ginst->vert_count), hipMemcpyHostToDevice));

    param_t* small = new param_t[ginst->vert_count];
    memset(small, -1, ginst->vert_count * sizeof(param_t));
    param_t* medium = new param_t[ginst->vert_count];
    memset(medium, -1, ginst->vert_count * sizeof(param_t));
    param_t* large = new param_t[ginst->vert_count];
    memset(large, -1, ginst->vert_count * sizeof(param_t));
    
    param_t* small_FQ;
    H_ERR(hipMalloc((void **)&small_FQ, sizeof(param_t)*(ginst->vert_count)));
    H_ERR(hipMemcpy(small_FQ, small, sizeof(param_t)*(ginst->vert_count), hipMemcpyHostToDevice));
    param_t* medium_FQ;
    H_ERR(hipMalloc((void **)&medium_FQ, sizeof(param_t)*(ginst->vert_count)));
    H_ERR(hipMemcpy(medium_FQ, medium, sizeof(param_t)*(ginst->vert_count), hipMemcpyHostToDevice));
    param_t* large_FQ;
    H_ERR(hipMalloc((void **)&large_FQ, sizeof(param_t)*(ginst->vert_count)));
    H_ERR(hipMemcpy(large_FQ, large, sizeof(param_t)*(ginst->vert_count), hipMemcpyHostToDevice));

    param_t* small_BU;
    H_ERR(hipMalloc((void **)&small_BU, sizeof(param_t)*(ginst->vert_count)));
    H_ERR(hipMemcpy(small_BU, small, sizeof(param_t)*(ginst->vert_count), hipMemcpyHostToDevice));
    param_t* medium_BU;
    H_ERR(hipMalloc((void **)&medium_BU, sizeof(param_t)*(ginst->vert_count)));
    H_ERR(hipMemcpy(medium_BU, medium, sizeof(param_t)*(ginst->vert_count), hipMemcpyHostToDevice));
    param_t* large_BU;
    H_ERR(hipMalloc((void **)&large_BU, sizeof(param_t)*(ginst->vert_count)));
    H_ERR(hipMemcpy(large_BU, large, sizeof(param_t)*(ginst->vert_count), hipMemcpyHostToDevice));

    param_t* Status = new param_t[ginst->vert_count];
    memset(Status, -1, ginst->vert_count * sizeof(param_t));
    Status[init] = 0;
    param_t* Status_cu;
    H_ERR(hipMalloc((void **)&Status_cu, sizeof(param_t)*(ginst->vert_count)));
    H_ERR(hipMemcpy(Status_cu, Status, sizeof(param_t)*(ginst->vert_count), hipMemcpyHostToDevice));
    
    param_t* small_cnt = (param_t*)malloc(sizeof(param_t));
    *small_cnt = 0;
    param_t* small_CNT;
    H_ERR(hipMalloc((void**)&small_CNT, sizeof(param_t)));
    H_ERR(hipMemcpy(small_CNT, small_cnt, sizeof(param_t), hipMemcpyHostToDevice));    
    param_t* medium_cnt = (param_t*)malloc(sizeof(param_t));
    *medium_cnt = 0;
    param_t* medium_CNT;
    H_ERR(hipMalloc((void**)&medium_CNT, sizeof(param_t)));
    H_ERR(hipMemcpy(medium_CNT, medium_cnt, sizeof(param_t), hipMemcpyHostToDevice));
    param_t* large_cnt = (param_t*)malloc(sizeof(param_t));
    *large_cnt = 0;
    param_t* large_CNT;
    H_ERR(hipMalloc((void**)&large_CNT, sizeof(param_t)));
    H_ERR(hipMemcpy(large_CNT, large_cnt, sizeof(param_t), hipMemcpyHostToDevice));

    param_t* level = (param_t*)malloc(sizeof(param_t));
    *level = 0;
    param_t* d_level;
    H_ERR(hipMalloc((void**)&d_level, sizeof(param_t)*1));
    H_ERR(hipMemcpy(d_level, level, sizeof(param_t)*1, hipMemcpyHostToDevice));
    //getchar(); 
    param_t* tag = new param_t[k];
    memset(tag, -1, k*sizeof(param_t));
    double T1 = 0, T2 = 0, T3 = 0, T4 = 0;
    double t1 = 0, t2 = 0, t3 = 0;
    double t4 = 0, t5 = 0, t6 = 0;
    double* T = new double[3];
    //for(int i = 0; i < 3; i++)
    //    T[i]=0;
    double naivetime=wtime();
    BFS_ControlFlow<param_t, index_t>(padding, dataset, init, adaptive, ratio, tag, iter, t1, t2, t3, t4, t5, t6, T1, T2, T3, T4, BlockDim, GridDim, status, status_cpp, Status, Status_cu, small, medium, large, small_FQ, medium_FQ, large_FQ, small_cnt, medium_cnt, large_cnt, small_CNT, medium_CNT, large_CNT, ginst->vert_count, level, d_level, ginst->beg_pos, beg_cu, csr_cu, alpha, small_BU, medium_BU, large_BU);
    hipDeviceSynchronize();
    cout<<endl<<"naivetime="<<(wtime()-naivetime)*1000 <<" ms"<<endl;
    double end = wtime(); 
    H_ERR(hipMemcpy(Status, Status_cu, sizeof(param_t)*ginst->vert_count, hipMemcpyDeviceToHost));
    std::cout<<"\nThe GPU-based status[]: \n";
    int mismatch=0;
    //assert(memcmp(Status, status, sizeof(param_t)*ginst->vert_count)==0);
    //std::cout<<"We finished<<<<<<<<<<<<<<<<<<\n"; 
    
    for(int i = 0; i < ginst->vert_count; i++) {
        if(Status[i]!=status[i]){
            mismatch++;
      //      std::cout<<Status[i]<<" ";     
    //        std::cout<<i<<" ";
        }
    }
    std::cout<<"\n\nCurrent level = "<<level[0]<<" "<<mismatch<<" padding="<<padding;
    std::cout<<"\nTime consumption of top-down && bottom up BFS: "<<T1+T2+T3+T4<<" seconds from the starting vertex "<<init<<"\n";
    file1<<dataset<<": ";
    for(int i=0; i<k; i++)
        file1<<tag[i]<<" ";
    file<<"\n";
    std::cout<<"\ntop-down FQ_gen: "<<T1<<" seconds. with counting: "<<t4<<", prefix-offset: "<<t5<<", FQ_gen: "<<t6<<" seconds.\n";
    std::cout<<"top-down SA update: "<<T2<<" seconds.\n";
    std::cout<<"bottom-up FQ_gen: "<<T3<<" seconds. with ballot_counting: "<<t1<<", prefix-offset: "<<t2<<", FQ_gen: "<<t3<<" seconds.\n";
    std::cout<<"bottom-up SA update: "<<T4<<" seconds.\n";
    std::cout<<"top-down: "<<T1+T2<<" seconds.\n";
    std::cout<<"bottom-up: "<<T3+T4<<" seconds.\n";
  
      //anil 1.20.2019
     ofstream Timefile("time_Baseline+adaptive+FQ+wb+async_P6000_Camera_more_synch.csv", ios::app);
      cout<<dataset<<";"<<"Total time of traversal (ms)="<<T1+T2+T3+T4<<endl;
Timefile<<dataset<<";"<<T1+T2<<";"<<T3+T4<<";"<<T1+T2+T3+T4<<endl;
     Timefile.close();
    //~1.20.2019

//std::cout<<"hub_level time consumption:"<<T[0]<<", "<<T[1]<<", "<<T[2]<<"\n";
    //file<<"This is scan-free version...\n";
    //file<<"Iteration time_consumption #frontiers with alpha="<<alpha<<" and total time: "<<T1+T2+T3+T4<<" and bottom-up FQ_gen: "<<T3<<"\n";
    //file<<init<<","<<ginst->vert_count-invalid<<","<<alpha<<","<<T1+T2<<","<<T1<<","<<T2<<","<<T1*100/(T1+T2)<<"%"<<endl<<endl;
    /*for(int i = 0; i < k; i++){
        if(tag[i]==0)
            file<<"TD_ratio:"<<ratio[i]<<" "<<iter[i]<<"\n";
        else if(tag[i]==1)
            file<<"BU_ratio:"<<ratio[i]<<" "<<iter[i]<<"\n";
    }*/
    //file<<filename<<","<<T1+T2+T3+T4<<","<<padding<<"\n";
    
    file.close();
    hipFree(Status_cu); 
    hipFree(small_FQ);
    hipFree(medium_FQ);
    hipFree(large_FQ);
    std::cout<<"Now we finish!\n";
 //   getchar();
    return 0;
}

